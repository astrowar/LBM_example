
#include "hip/hip_runtime.h"
#include "lbm_gpu.h"

#define Qcc (19)
void   D3Q19( real* ex, real* ey, real* ez, int* oppos, real* wt );

LBMGrid::~LBMGrid()
{
    hipFree( f );
    hipFree( feq );
    hipFree( f_new );

    hipFree( rho );
    hipFree( ux );
    hipFree( uy );
    hipFree( uz );

    hipFree( sigma );
    hipFree( ex );
    hipFree( ey );
    hipFree( ez );
    hipFree( oppos );
    hipFree( wt );
}
LBMGrid::LBMGrid( size_t _nx, size_t _ny, size_t _nz ) {
    
    this->nx = _nx;
    this->ny = _ny;
    this->nz = _nz;

    hipMalloc( &f, nx * ny * nz * Qcc * sizeof( real ) );
    hipMalloc( &feq, nx * ny * nz * Qcc * sizeof( real ) );
    hipMalloc( &f_new, nx * ny * nz * Qcc * sizeof( real ) );
    
     
   //  f = new real[NX * NY * NZ * Qcc];
   //   feq = new real[NX * NY * NZ * Qcc];
   // f_new = new real[NX * NY * NZ * Qcc];


    hipMalloc( &rho, nx * ny * nz *  sizeof( real ) );
    hipMalloc( &ux, nx * ny * nz * sizeof( real ) );
    hipMalloc( &uy, nx * ny * nz *  sizeof( real ) );
    hipMalloc( &uz, nx * ny * nz *  sizeof( real ) );

    // density and velocity
    // rho = new real[NX * NY * NZ];
    //  ux = new real[NX * NY * NZ];
   //  uy = new real[NX * NY * NZ];
    //uz = new real[NX * NY * NZ];

     cpu_ux = new real[nx * ny * nz];
     cpu_uy = new real[nx * ny * nz];
     cpu_uz = new real[nx * ny * nz];

    // rate-of-strain
    hipMalloc( &sigma, nx * ny * nz * sizeof( real ) );
   // sigma = new real[NX * NY * NZ];


    hipMalloc( &ex, Qcc * sizeof( real ) );
    hipMalloc( &ey, Qcc * sizeof( real ) );
    hipMalloc( &ez, Qcc * sizeof( real ) );
    hipMalloc( &oppos, Qcc * sizeof( int ) );
    hipMalloc( &wt, Qcc * sizeof( real ) );

    // D3Q9 parameters
    // ex = new real[Qcc];
    //  ey = new real[Qcc];
    //  ez = new real[Qcc];
    //  oppos = new int[Qcc];
    // wt = new real[Qcc];


 
    D3Q19( ex, ey, ez, oppos, wt );

}


 